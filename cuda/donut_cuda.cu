

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
//#include <unistd.h>
#include <chrono>
#include <thread>
#include <cstring>



//define the donut 
#define R1 (1)
#define R2 (2)

//define the render grid
#define thetaStep (0.06)
#define phiStep (0.03)

//define the camera
#define K1 (30) //focus(origin) to the screen
#define resH (22) //render resolution
#define resW (80)

//define the pos
#define K2 (5) //focus(origin) to the donut center

//define the light, a opposite vecter of the light beem, simulating an infinite plane light source
//it should be a unit vector
//pls notice that it can not render the shade
#define lX (0)
#define lY (0.707)
#define lZ (-0.707)

//define buf depth for oozBuf and lumBuf, bigger if using smaller grid
#define BUF_DEPTH (200)




__global__ void calcPointCuda(const float A, const float B, float* oozBuf, float* lumBuf, int* depthBuf){

    //get the theta and phi from index
    float theta = blockIdx.x * thetaStep;
    float phi = threadIdx.x * phiStep;

    float sinTheta = sinf(theta);
    float cosTheta = cosf(theta);

    float sinPhi = sinf(phi);
    float cosPhi = cosf(phi);

    //common value
    float sinA = sinf(A);
    float cosA = cosf(A);
    float sinB = sinf(B);
    float cosB = cosf(B);


    //without rotations, z = 0
    float x = R2 + R1 * cosTheta;
    float y = R1 * sinTheta;

    //rotate matrix
    float xRotatedo = x * (cosB * cosPhi + sinA * sinB * sinPhi) - y * cosA * sinB;
    float yRotatedo = x * (sinB * cosPhi - sinA * cosB * sinPhi) + y * cosA * cosB;

    float zRotated = K2 + cosA * x * sinPhi + y * sinA;

    float ooz = 1 / zRotated;

    int xP = (int) (resW / 2 + K1 * ooz * xRotatedo);
    int yP = (int) (resH / 2 - K1 * 0.5 * ooz * yRotatedo);

    if(xP >= resW || xP < 0 || yP >= resH || yP < 0){
        //out of the screen, end thread for the point
        return;
    }

    //illumination, Lvec * Normalvec
    float lum = lX * (cosTheta * cosPhi * cosB + 
                sinPhi * cosTheta * sinA * sinB - 
                sinTheta * cosA * sinB) +
                lY * (cosTheta * cosPhi * sinB +
                cosB * sinTheta * cosA -
                cosB * sinPhi * cosTheta * sinA) +
                lZ * (sinA * sinTheta + sinPhi * cosTheta * cosA);

    int index = yP * resW + xP;
    int indexBuf = BUF_DEPTH * index;
    //take one slot for the point
    int depth = atomicAdd(depthBuf+index, 1);
    //if(depth > BUF_DEPTH)printf("%d\n", depth);
    //store the ooz and lum
    oozBuf[indexBuf + depth] = ooz;
    lumBuf[indexBuf + depth] = lum;


}


__global__ void renderPixCuda(float* oozBuf, float* lumBuf, int* depthBuf, char* output){
    //results from previous kernel
    int x = threadIdx.x;
    int y = blockIdx.x;

    int index = y * resW + x;
    int indexBuf = BUF_DEPTH * index;

    float oozMax = 0;
    

    for(int i = 0; i < depthBuf[index]; i++){
        //iterate each corresponding point
        float ooz = oozBuf[indexBuf + i];
        float lum = lumBuf[indexBuf + i];

        if(ooz > oozMax){/* current point is closer to the viewer */
            oozMax = ooz;
            if(lum > 0 ){/* the point is visible */
                int lumIndex = lum * 11.3; //map the illuminance to the index
                output[index] = ".,-~:;=!*#$@"[lumIndex];
            }else{
                output[index] = ' ';
            }
        }
    }

}



__host__ int main(){

    const size_t outputSize = resH * resW * sizeof(char);
    char outputF[outputSize]; //host buf 

    
    char* output; //device buf
    if(hipSuccess != hipMalloc(&output, outputSize)){
        //fail
        printf("[!]Unable to aloocate the output frame.");
        exit(-1);
    }

    float* oozBuf;
    if(hipSuccess != hipMalloc(&oozBuf, BUF_DEPTH * resH * resW * sizeof(float))){
        //fail
        printf("[!]Unable to aloocate the oozBuf frame.");
        exit(-1);
    }

    float* lumBuf;
    if(hipSuccess != hipMalloc(&lumBuf, BUF_DEPTH * resH * resW * sizeof(float))){
        //fail
        printf("[!]Unable to aloocate the lumBuf frame.");
        exit(-1);
    }

    int* depthBuf;
    if(hipSuccess != hipMalloc(&depthBuf, resH * resW * sizeof(int))){
        //fail
        printf("[!]Unable to aloocate the depthBuf frame.");
        exit(-1);
    }


    //by default(A = 0), the donut is horizontally positioned, the hole will be invisible at the beginning
    float A = 3.14/2, B = 0; 
    
    while(true){
        //clear the buf on the gpu
        hipMemset(output, ' ', resW * resH * sizeof(char));
        hipMemset(oozBuf, 0, BUF_DEPTH * resH * resW * sizeof(float));
        hipMemset(lumBuf, 0, BUF_DEPTH * resH * resW * sizeof(float));
        hipMemset(depthBuf, 0, resH * resW * sizeof(int));
        
        //thread for every point on the donut, grid for small circle, block for big circle
        calcPointCuda<<<6.28/thetaStep,6.28/phiStep>>>(A, B, oozBuf, lumBuf, depthBuf);

        hipDeviceSynchronize();    
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("[!]CUDA error:calcpoint: %s\n", hipGetErrorString(error));
            exit(1);
        }

        renderPixCuda<<<resH,resW>>>(oozBuf, lumBuf, depthBuf, output);

        hipDeviceSynchronize();    
        error = hipGetLastError();
        if (error != hipSuccess) {
            printf("[!]CUDA error:renderpix: %s\n", hipGetErrorString(error));
            exit(1);
        }
        //get the frame from gpu
        hipMemcpy(outputF, output, outputSize, hipMemcpyDeviceToHost);


        //output
        printf("\x1b[H");
        for (int k = 0; k < resH * resW + 1; k++) {
            putchar(k % resW ? outputF[k] : 10); //newline or char
        }

        //control the rotation speed
         A += 0.04;
         B += 0.02;
        std::this_thread::sleep_for(std::chrono::milliseconds(30));
    }

    

}






