

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
//#include <unistd.h>
#include <chrono>
#include <thread>
#include <cstring>

//define the donut 
const float R1 = 1;
const float R2 = 2;

//define the render grid
const float thetaStep = 0.07;
const float phiStep = 0.02;

//define the camera
const float K1 = 30; //focus(origin) to the screen
const int resH = 22; //render resolution
const int resW = 80;

//define the pos
const float K2 = 5; //focus(origin) to the donut center

//define the light, a opposite vecter of the light beem, simulating an infinite plane light source
//it should be a unit vector
const float lX = 0;
const float lY = 0.707;
const float lZ = -0.707;



__host__ void renderFrame(float A, float B, char* outputP){ //A: rotate about x-axis, B: rotate about z-axis

    static float zBuf[resW][resH]; 

    memset(outputP, ' ', resW * resH * sizeof(char));
    memset(zBuf, 0, resW * resH * sizeof(float));

    float sinA = sin(A), sinB = sin(B), cosA = cos(A), cosB = cos(B);

    //iterate the donut for one frame
    for(float theta = 0; theta < 3.14 * 2; theta += thetaStep){ //the smaller circle

        float sinTheta = sin(theta), cosTheta = cos(theta);

        for(float phi = 0; phi < 3.14 * 2; phi += phiStep){ //the larger circle
            float sinPhi = sin(phi), cosPhi = cos(phi);
            
            //for every point with theta, phi

            //without rotations, z = 0
            float x = R2 + R1 * cosTheta;
            float y = R1 * sinTheta;

            //rotate matrix
            float xRotatedo = x * (cosB * cosPhi + sinA * sinB * sinPhi) - y * cosA * sinB;
            float yRotatedo = x * (sinB * cosPhi - sinA * cosB * sinPhi) + y * cosA * cosB;

            // float xRotated = cosPhi * x * cosB - sinB * (sinPhi * x * cosA - sinTheta * sinA);
            // float yRotated = cosPhi * x * sinB + cosB * (sinPhi * x * cosA - sinTheta * sinA);

            float zRotated = K2 + cosA * x * sinPhi + y * sinA;

            float ooz = 1 / zRotated;

            int xP = (int) (resW / 2 + K1 * ooz * xRotatedo);
            int yP = (int) (resH / 2 - K1 * 0.5 * ooz * yRotatedo);

            if(xP >= resW || xP < 0 || yP >= resH || yP < 0){
                //out of the screen, pass
                continue;
            }

            //illumination, Lvec * Normalvec
            float lum = lX * (cosTheta * cosPhi * cosB + 
                        sinPhi * cosTheta * sinA * sinB - 
                        sinTheta * cosA * sinB) +
                        lY * (cosTheta * cosPhi * sinB +
                        cosB * sinTheta * cosA -
                        cosB * sinPhi * cosTheta * sinA) +
                        lZ * (sinA * sinTheta + sinPhi * cosTheta * cosA);

            if( ooz > zBuf[xP][yP] /* current point is closer to the viewer */&&
                lum > 0 /* the point is visible */){
                zBuf[xP][yP] = ooz;

                int lumIndex = lum * 11.3; //map the illuminance to the index
                outputP[xP + resW * yP] = ".,-~:;=!*#$@"[lumIndex];
            }


        }

    } 



}


__global__ void renderPointCuda(const float A, const float B, float* zBuf, char* output){

    //get the theta and phi from index
    float theta = blockIdx.x * thetaStep;
    float phi = threadIdx.x * phiStep;

    float sinTheta = sinf(theta);
    float cosTheta = cosf(theta);

    float sinPhi = sinf(phi);
    float cosPhi = cosf(phi);

    //common value
    float sinA = sinf(A);
    float cosA = cosf(A);
    float sinB = sinf(B);
    float cosB = cosf(B);


    //without rotations, z = 0
    float x = R2 + R1 * cosTheta;
    float y = R1 * sinTheta;

    //rotate matrix
    float xRotatedo = x * (cosB * cosPhi + sinA * sinB * sinPhi) - y * cosA * sinB;
    float yRotatedo = x * (sinB * cosPhi - sinA * cosB * sinPhi) + y * cosA * cosB;

    float zRotated = K2 + cosA * x * sinPhi + y * sinA;

    float ooz = 1 / zRotated;

    int xP = (int) (resW / 2 + K1 * ooz * xRotatedo);
    int yP = (int) (resH / 2 - K1 * 0.5 * ooz * yRotatedo);

    if(xP >= resW || xP < 0 || yP >= resH || yP < 0){
        //out of the screen, end thread for the point
        return;
    }

    //illumination, Lvec * Normalvec
    float lum = lX * (cosTheta * cosPhi * cosB + 
                sinPhi * cosTheta * sinA * sinB - 
                sinTheta * cosA * sinB) +
                lY * (cosTheta * cosPhi * sinB +
                cosB * sinTheta * cosA -
                cosB * sinPhi * cosTheta * sinA) +
                lZ * (sinA * sinTheta + sinPhi * cosTheta * cosA);

    if( ooz > zBuf[xP + yP * resW] /* current point is closer to the viewer */&&
        lum > 0 /* the point is visible */){
        zBuf[xP + yP * resW] = ooz;

        int lumIndex = lum * 11.3; //map the illuminance to the index
        output[xP + resW * yP] = ".,-~:;=!*#$@"[lumIndex];
    }



}


__host__ int main(){

    const size_t outputSize = resH * resW * sizeof(char);
    char outputF[outputSize];

    
    char* output;
    if(hipSuccess != hipMalloc(&output, outputSize)){
        //fail
        printf("[!]Unable to aloocate the output frame.");
        exit(-1);
    }

    float* zBuf;
    if(hipSuccess != hipMalloc(&zBuf, resH * resW * sizeof(float))){
        //fail
        printf("[!]Unable to aloocate the zBuf frame.");
        exit(-1);
    }


    //by default(A = 0), the donut is horizontally positioned, the hole will be invisible at the beginning
    float A = 3.14/2, B = 0; 
    
    while(true){
        //clear the buf on the gpu
        hipMemset(output, ' ', resW * resH * sizeof(char));
        hipMemset(zBuf, 0, resW * resH * sizeof(float));
        
        //thread for every point on the donut, grid for small circle, block for big circle
        renderPointCuda<<<6.28/thetaStep,6.28/phiStep>>>(A, B, zBuf, output);

        hipDeviceSynchronize();    
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("[!]CUDA error: %s\n", hipGetErrorString(error));
            exit(1);
        }
        
        //get the frame from gpu
        hipMemcpy(outputF, output, outputSize, hipMemcpyDeviceToHost);


        //output
        printf("\x1b[H");
        for (int k = 0; k < resH * resW + 1; k++) {
            putchar(k % resW ? outputF[k] : 10); //newline or char
        }

        //control the rotation speed
         A += 0.04;
         B += 0.02;
        std::this_thread::sleep_for(std::chrono::milliseconds(30));
    }

    

}






